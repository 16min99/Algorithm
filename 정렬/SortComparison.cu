#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
using namespace std;

#include <hip/hip_runtime.h>       
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define THREADS pow(2,10)// 1024, ������ �� 
#define BLOCKS pow(2,17) // �����带 ��� ���� ����� ��
#define NUM_VALS THREADS*BLOCKS // ��� * ������ = �����ͼ� ũ�� (2^n)  

void swap(int arr[], int a, int b);
void QuickSort(int a[], int l, int r);
void InsertionSort(int a[], int front, int back, int gap);
void ShellSort(int a[], int n);
void checkSort(int a[], int n);
__global__ void bitonic_sort_step(int* dev_values, int j, int k);
void BitonicSort(int* values);

int main() {
	cout << "�����ͼ� ũ�� : " << NUM_VALS << endl;
	cout << "������ �� : " << THREADS << endl;
	int* arr1 = new int[NUM_VALS]; // �����Ҵ�
	int* arr2 = new int[NUM_VALS]; // �����Ҵ�
	int* arr3 = new int[NUM_VALS]; // �����Ҵ�


	srand(time(NULL));
	for (int i = 0; i < NUM_VALS; i++)
	{
		arr1[i] = rand() % (int)pow(2, 20); // ���� ��
		arr2[i] = arr1[i]; // �迭 ���Ұ� ����
		arr3[i] = arr2[i]; // �迭 ���Ұ� ����
	}


	//�ð� 
	clock_t start, end;
	//start = clock();
	//QuickSort(arr1, 0, NUM_VALS - 1); // �� ��Ʈ
	//end = clock();
	//cout << "Median-of-three Quick sort elapsed time: : " << end - start << "ms\n";
	//cout << "Median-of-three Quick sort is ";
	//checkSort(arr1, NUM_VALS);

	//start = clock();
	//ShellSort(arr2, NUM_VALS); // �� ��Ʈ
	//end = clock();
	//cout << "Shell sort elapsed time: : " << end - start << "ms\n";
	//cout << "Shell sort is ";
	//checkSort(arr2, NUM_VALS);

	start = clock();
	BitonicSort(arr3); // �� ��Ʈ
	end = clock();
	cout << "Bitonic sort used GUDA elapsed time: : " << end - start << "ms\n";
	cout << "Bitonic sort used GUDA sort is ";
	checkSort(arr3, NUM_VALS);

	cout << "12161783 ���α�";

}



//�迭�� �ּҸ� �޾ƿ��� ������ ���� �ٲ� �� �ִ�.
void swap(int arr[], int a, int b) {
	int temp = arr[a];
	arr[a] = arr[b];
	arr[b] = temp;
}

void QuickSort(int a[], int l, int r) {
	int i, j, m, v; // v is pivot
	if (r - l > 1) {
		m = (l + r) / 2;
		/*
		�߰����� �̿��� �������
		l,m,r�� ������ �켱������ ������ ����
		*/
		if (a[l] > a[m]) swap(a, l, m);
		if (a[l] > a[r]) swap(a, l, r);
		if (a[m] > a[r]) swap(a, m, r);
		swap(a, m, r - 1);  // m( l,m,r�� �߰���)�� pivot�ڸ� (r-1)�� �ű� r�ڸ��� lmr�� �ִ밪
							// �̶� ����Ʈ �˰����� ���ȣ��� ���ؼ� r�ڸ��� �ᱹ �ִ밪�� ���ϰ� ��

		v = a[r - 1]; i = l; j = r - 1;

		for (; ; ) {
			while (a[++i] < v);
			while (a[--j] > v);
			if (i >= j) break;//i��j�� �ѱ�� 
			swap(a, i, j);
		}
		swap(a, i, r - 1); // pivot�� i�� �ٲ۴�
		//i�� ����(����pivot)���� ��, �� ������ ���ȣ��
		QuickSort(a, l, i - 1);
		QuickSort(a, i + 1, r);
	}
	else if (a[l] > a[r])
		swap(a, l, r);
}

void InsertionSort(int a[], int front, int back, int gap) {
	for (int i = front + gap; i <= back; i += gap) {
		int j;
		int tmp = a[i]; // ��ġ�� ã�� 
		for (j = i - gap; (j >= front) && (a[j] > tmp); j -= gap) {
			a[j + gap] = a[j]; // gap ��ŭ �ٰ� 
		}
		a[j + gap] = tmp;
	}
}

void ShellSort(int a[], int n)
{
	int h, v;
	int gap[1000]; //gap�� ��Ƶδ� �迭 

	gap[0] = 1;
	int gap_idx = 0;//gap�ε���
	while (gap[gap_idx] < n) {
		gap_idx++;
		gap[gap_idx] = pow(4, gap_idx) + 3 * pow(2, gap_idx - 1) + 1; // gap ũ�⸦ ������ �ۼ� ����
	}


	for (; gap_idx >= 0; gap_idx--) {
		for (int i = 0; i < gap[gap_idx]; i++) {

			if (i + gap[gap_idx] < n) {
				InsertionSort(a, i, n - 1, gap[gap_idx]); //gap �������� �������� ����
			}
		}
	}
}

void checkSort(int a[], int n) {
	int i, sorted;
	sorted = true;
	for (i = 0; i < n - 1; i++)
	{
		if (a[i] > a[i + 1])
		{
			sorted = false;
		}
		if (!sorted)
		{
			break;
		}
	}
	if (sorted)
	{
		cout << "Sorting complete!" << endl;
	}
	else
	{
		cout << "Error during sorting..." << endl;
	}
}


__global__ void bitonic_sort_step(int* dev_values, int j, int k)
{
	unsigned int i, ixj; // ������ �з��� ���� ����
	i = threadIdx.x + blockDim.x * blockIdx.x;//blockIdx.x ���纻 ����

	ixj = i ^ j;

	// ������ ���� ����
	if ((ixj) > i) {
		if ((i & k) == 0) {
			// �������� ���� 
			if (dev_values[i] > dev_values[ixj]) {
				int temp = dev_values[i];	
				dev_values[i] = dev_values[ixj];
				dev_values[ixj] = temp;
			}
		}
		if ((i & k) != 0) {
			// �������� ����
			if (dev_values[i] < dev_values[ixj]) {
				int temp = dev_values[i];
				dev_values[i] = dev_values[ixj];
				dev_values[ixj] = temp;
			}
		}
	}
}


void BitonicSort(int* values) {
	int* dev_values;
	size_t size = NUM_VALS * sizeof(int);

	hipMalloc((void**)&dev_values, size);
	hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

	// <<<>>>>�� ù��° ���ڿ� 3�������� �������� �̰��� 2�����ε� 1�̹Ƿ�
	// BLOCKS*1 =BLOCKS����ŭ��
	dim3 blocks(BLOCKS, 1);    /* Number of blocks   */
	dim3 threads(THREADS, 1);  /* Number of threads  */

	int j, k;
	/* Major step */
	for (k = 2; k <= NUM_VALS; k <<= 1) {
		/* Minor step */
		for (j = k >> 1; j > 0; j = j >> 1) {
			bitonic_sort_step << <blocks, threads >> > (dev_values, j, k);
			//blocks ��ŭ ���纻 ����(�Ҵ�) �ǰ�, ���� block�� Threads���� �����尡 ������
		}
	}
	hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
	hipFree(dev_values);
}